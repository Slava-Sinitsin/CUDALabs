#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include <cstdio>
#include <cstdlib>
#include <ctime>

void filling(double *a, int n) {
    for (int i = 0; i < n * n; i++) {
        a[i] = rand() % 9 + 1;
    }
}

__global__ void div(double *a, const int *N, const int *shift, const int *level) {
    int i = threadIdx.x;
    int j = *N - threadIdx.y - 1;
    a[i * (*N) + j + (*shift) + i * (*level)] /= a[i * (*N) + (*shift) + i * (*level)];
}

__global__ void sub(double *a, const int *N, const int *shift, const int *level) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    a[(i + 1) * (*N) + (*shift) + (i + 1) * (*level) + j] -= a[(*shift) + j];
}

__global__ void mult(double *a, int *N, const int *shift, const int *level, double *answer) {
    for (int i = 0; i < *N; i++) {
        (*answer) *= a[i * (*N) + (*shift) + i * (*level)];
    }
}

double detCalc(double *a, int N) {
    double answer = 1;
    int shift = 0;
    int dailySize = 0;
    double *a_d;
    int *N_d;
    int *shift_d;
    int *level_d;
    double *answer_d;
    hipMalloc((void **) &a_d, N * N * sizeof(double));
    hipMalloc((void **) &N_d, sizeof(int));
    hipMalloc((void **) &shift_d, sizeof(int));
    hipMalloc((void **) &level_d, sizeof(int));
    hipMalloc((void **) &answer_d, sizeof(double));
    hipMemcpy(a_d, a, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(answer_d, &answer, sizeof(double), hipMemcpyHostToDevice);
    for (int level = 0; level < N; level++) {
        dailySize = N - level;
        shift = level * (N + 1);
        hipMemcpy(N_d, &dailySize, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(shift_d, &shift, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(level_d, &level, sizeof(int), hipMemcpyHostToDevice);
        mult <<<1, dim3(1, 1, 1) >>>(a_d, N_d, shift_d, level_d, answer_d);
        hipDeviceSynchronize();
        div <<<8, dim3(dailySize, dailySize, 1) >>>(a_d, N_d, shift_d, level_d);
        hipDeviceSynchronize();
        sub <<<8, dim3(dailySize - 1, dailySize, 1) >>>(a_d, N_d, shift_d, level_d);
        hipDeviceSynchronize();
    }
    hipMemcpy(&answer, answer_d, sizeof(double), hipMemcpyDeviceToHost);
    hipFree(a_d);
    hipFree(N_d);
    hipFree(shift_d);
    hipFree(answer_d);
    return answer;
}

double getTime(double *a, int n) {
    clock_t bg = clock();
    detCalc(a, n);
    double time_go = clock() - bg;
    time_go /= 1000;
    return time_go;
}

void getInfo() {
    hipDeviceProp_t deviceProp{};
    hipGetDeviceProperties(&deviceProp, 0);
    printf("Device name: %s\n", deviceProp.name);
    printf("Total global memory: %ull\n", deviceProp.totalGlobalMem);
    printf("Shared memory per block: %d\n", deviceProp.sharedMemPerBlock);
    printf("Registers per block: %d\n", deviceProp.regsPerBlock);
    printf("Warp size: %d\n", deviceProp.warpSize);
    printf("Memory pitch: %d\n", deviceProp.memPitch);
    printf("Max threads per block: %d\n", deviceProp.maxThreadsPerBlock);
    printf("Max threads dimensions: x = %d, y = %d, z = %d\n",
           deviceProp.maxThreadsDim[0],
           deviceProp.maxThreadsDim[1],
           deviceProp.maxThreadsDim[2]);
    printf("Max grid size: x = %d, y = %d, z = %d\n",
           deviceProp.maxGridSize[0],
           deviceProp.maxGridSize[1],
           deviceProp.maxGridSize[2]);
    printf("Clock rate: %d\n", deviceProp.clockRate);
    printf("Total constant memory: %d\n", deviceProp.totalConstMem);
    printf("Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
    printf("Texture alignment: %d\n", deviceProp.textureAlignment);
    printf("Device overlap: %d\n", deviceProp.deviceOverlap);
    printf("Multiprocessor count: %d\n", deviceProp.multiProcessorCount);
    printf("Kernel execution timeout enabled: %s\n", deviceProp.kernelExecTimeoutEnabled ? "true" : "false");
}

void speedTest(int start, int finish, int step) {
    auto *timeToFile = (double *) malloc((((finish - start) / step) * sizeof(double)));
    int k = 0;
    for (int i = start; i < (finish - start) / step; i += step) {
        auto *a = (double *) malloc((i + 1) * (i + 1) * sizeof(double));
        filling(a, i + 1);
        timeToFile[k++] = getTime(a, i + 1);
        free(a);
        //printf("%f\n", timeToFile[i]);
    }
    FILE *fp;
    timeToFile[0] = 0;
    if ((fp = fopen("write.txt", "w")) != nullptr) {
        for (int i = 0; i < (finish - start) / step; ++i) {
            fprintf(fp, "%.6f\n", timeToFile[i]);
        }
    }
    fclose(fp);
    free(timeToFile);
}

int main() {
    int start = 0;
    int finish = 500;
    int step = 1;
    getInfo();
    printf("%d %d %d", start, finish, step);
    speedTest(start, finish, step);
    printf("\nFinish");
    return 0;
}